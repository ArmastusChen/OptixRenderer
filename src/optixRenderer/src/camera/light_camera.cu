#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "random.h"

using namespace optix;


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        cameraU, , );
rtDeclareVariable(float3,        cameraV, , );
rtDeclareVariable(float3,        cameraW, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(int,           max_depth, , ); 

rtBuffer<float3, 2>              intensity_buffer;

rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(int, cameraMode, , );
rtDeclareVariable(unsigned int, initSeed, , ); 

rtDeclareVariable(unsigned int, envWidth, , );
rtDeclareVariable(unsigned int, envHeight, , );


RT_PROGRAM void pinhole_camera()
{
    size_t2 screen = make_size_t2(160, 120);
    
    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;

    unsigned int seed = tea<32>( 
        ( (initSeed)*(screen.x*launch_index.y+launch_index.x) + initSeed ), 
        ( (screen.y * launch_index.x + launch_index.y) * initSeed ) ); 
    
    // Presampling to get the normal and the origins
    float3 normalAccu = make_float3(0.0);
    unsigned int sqrt_pre_samples = 2;
    unsigned int pre_samples_num = sqrt_pre_samples * sqrt_pre_samples;
    do{
        float3 ray_origin = eye;
        unsigned int x = pre_samples_num%sqrt_pre_samples;
        unsigned int y = pre_samples_num/sqrt_pre_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed) );
        float2 d = pixel + jitter* (inv_screen / sqrt_pre_samples );

        float3 ray_direction;
        ray_direction = normalize(d.x*cameraU + d.y*cameraV + cameraW);

        // Initialze per-ray data
        PerRayData_radiance prd;
        prd.attenuation = make_float3(1.f);
        prd.radiance = make_float3(0.f);
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;
        prd.direction = ray_direction;
        prd.origin = ray_origin; 
        prd.normal = make_float3(0.0);
        
        prd.brdfDirection = make_float3(1.0); 
        prd.isHitArea = false;
        prd.areaRadiance = make_float3(0.0);
        prd.areaDirection = make_float3(1.0); 
        prd.isHitEnv = false;
        prd.envRadiance = make_float3(0.0);
        prd.envDirection = make_float3(1.0);  

        Ray ray(ray_origin, ray_direction, 0, scene_epsilon);
        rtTrace(top_object, ray, prd); 
        normalAccu = normalAccu + prd.normal;
    } while(--pre_samples_num );

    // Sample pixel using jittering 
    if(normalAccu.x != 0 || normalAccu.y != 0 || normalAccu.z != 0){
        float3 upAxis = normalize(cameraV );
        float3 zAxis = normalize(normalAccu ); 
        float3 yAxis = upAxis - dot(upAxis, zAxis) * zAxis; 
        yAxis = normalize(yAxis );
        float3 xAxis = cross(zAxis, yAxis );  

        do{
            float3 ray_origin = eye;
            unsigned int x = samples_per_pixel%sqrt_num_samples;
            unsigned int y = samples_per_pixel/sqrt_num_samples;
            float2 jitter = make_float2(x-rnd(seed), y-rnd(seed) );
            float2 d = pixel + jitter*jitter_scale;
    
            float3 ray_direction;
            ray_direction = normalize(d.x*cameraU + d.y*cameraV + cameraW);
    
            // Initialze per-ray data
            PerRayData_radiance prd;
            prd.attenuation = make_float3(1.f);
            prd.radiance = make_float3(0.f);
            prd.done = false;
            prd.seed = seed;
            prd.depth = 0;
            prd.direction = ray_direction;
            prd.origin = ray_origin; 
            prd.normal = make_float3(0.0); 
            
            prd.brdfDirection = make_float3(1.0); 
            prd.isHitArea = false;
            prd.areaRadiance = make_float3(0.0);
            prd.areaDirection = make_float3(1.0); 
            prd.isHitEnv = false;
            prd.envRadiance = make_float3(0.0);
            prd.envDirection = make_float3(1.0);  
    
            // Each iteration is a segment of the ray path.  The closest hit will
            // return new segments to be traced here.
            for(;;) {
                Ray ray(ray_origin, ray_direction, 0, scene_epsilon);
                rtTrace(top_object, ray, prd);
     
                prd.depth++; 
            
                // Hit the light source or exceed the max depth
                if(prd.done || prd.depth >= max_depth) 
                    break; 
                
                // Update ray data for the next path segment
                ray_origin = prd.origin; 
                ray_direction = prd.direction;
                ray_origin = ray_origin + 0.1 * scene_epsilon * ray_direction;
            } 
    
            if(prd.depth >= 2 ){   
                if(true ){
                    float theta = acos(dot(prd.brdfDirection, zAxis ) );
                    float phi = atan2(
                        dot(prd.brdfDirection, yAxis), 
                        dot(prd.brdfDirection, xAxis ) );
                
                    if(phi > M_PIf )
                        phi = phi - 2 * M_PIf; 
        
                    float thetaId = (theta / M_PIf * 2) * envHeight;
                    float phiId = ( (phi / M_PIf ) + 1) * 0.5 * envWidth; 
                        
                    if(thetaId <= envHeight + 2){
                        if(thetaId >= envHeight) thetaId = envHeight-1e-6;
                        if(thetaId < 0 ) thetaId = 0;
                        if(phiId >= envWidth ) phiId = envWidth - 1e-6; 
                        if(phiId < 0 ) phiId = 0;
            
                        unsigned x = launch_index.x * envWidth + floor(phiId );
                        unsigned y = launch_index.y * envHeight + floor(thetaId );

                        intensity_buffer[make_uint2(x, y) ] += prd.radiance;
                    }
                }
                
                if(prd.isHitEnv ){
                    float theta = acos(dot(prd.envDirection, zAxis ) );
                    float phi = atan2(
                        dot(prd.envDirection, yAxis), 
                        dot(prd.envDirection, xAxis ) );
                    if(phi > M_PIf )
                        phi = phi - 2 * M_PIf; 
        
                    float thetaId = ( theta / M_PIf * 2) * envHeight;
                    float phiId = ( (phi / M_PIf ) + 1) * 0.5 * envWidth; 

                    if(thetaId <= envHeight + 2){
                        if(thetaId >= envHeight) thetaId = envHeight-1e-6;
                        if(thetaId < 0 ) thetaId = 0;
                        if(phiId >= envWidth ) phiId = envWidth - 1e-6; 
                        if(phiId < 0 ) phiId = 0;
            
                        unsigned x = launch_index.x * envWidth + floor(phiId );
                        unsigned y = launch_index.y * envHeight + floor(thetaId );

                        intensity_buffer[make_uint2(x, y) ] += prd.envRadiance; 
                    }
                }
    
                if(prd.isHitArea ){
                    float theta = acos(dot(prd.areaDirection, zAxis ) );
                    float phi = atan2(
                        dot(prd.areaDirection, yAxis), 
                        dot(prd.areaDirection, xAxis ) );
                    if(phi > M_PIf )
                        phi = phi - 2 * M_PIf; 
        
                    float thetaId = (theta / M_PIf * 2) * envHeight;
                    float phiId = ( (phi / M_PIf ) + 1) * 0.5 * envWidth; 

                    if(thetaId <= envHeight + 2){
                        if(thetaId >= envHeight) thetaId = envHeight-1e-6;
                        if(thetaId < 0 ) thetaId = 0;
                        if(phiId >= envWidth ) phiId = envWidth - 1e-6; 
                        if(phiId < 0 ) phiId = 0;
            
                        unsigned x = launch_index.x * envWidth + floor(phiId );
                        unsigned y = launch_index.y * envHeight + floor(thetaId );

                        intensity_buffer[make_uint2(x, y) ] += prd.areaRadiance;
                    }
                }
            }
        } while (--samples_per_pixel); 
    } 
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
}
