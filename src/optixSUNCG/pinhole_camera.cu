#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>
#include "helpers.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace_shadow{
    bool inShadow;
};


struct PerRayData_pathtrace{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

// The parameter of the camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        plight, , );
rtDeclareVariable(float,         lightIntensity, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(float,         infiniteFar, , );

rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float4, 2>              float_buffer;
rtBuffer<float, 2>               envcdfV;
rtBuffer<float, 2>               envcdfH;
rtBuffer<float, 2>               envmapPdf;

rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int, pathtrace_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(unsigned int, bounce_num, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Add the phong model for practice
rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(float, F0, , );
rtDeclareVariable(float3, specularColor, , );
rtDeclareVariable(int,    mode, , );

// Environment map background
rtTextureSampler<float4, 2> envmap;

// Spatially varying BRDF
rtTextureSampler<float4, 2> albedoMap;
rtTextureSampler<float4, 2> normalMap;
rtTextureSampler<float4, 2> roughMap;


/********************************************/
/**** Function to sample environment map ****/
/********************************************/
RT_CALLABLE_PROGRAM float3 EnvUVToDirec(float u, float v){ 
    // Turn uv coordinate into direction
    float theta = 2 * (u - 0.5) * M_PIf;
    float phi = M_PIf * (1 - v); 
    return make_float3(
                sinf(phi) * sinf(theta),
                cosf(phi),
                sinf(phi) * cosf(theta)
            );
}

RT_CALLABLE_PROGRAM float2 EnvDirecToUV(const float3& direc){ 
    float theta = atan2f( direc.x, direc.z );
    float phi = M_PIf - acosf(direc.y );
    float u = theta * (0.5f * M_1_PIf) + 0.5;
    if(u > 1)
        u = u-1;
    float v     = phi / M_PIf;
    return make_float2(u, v);
}

RT_CALLABLE_PROGRAM float EnvDirecToPdf(const float3& direc){
    float2 uv = EnvDirecToUV(direc);
    size_t2 pdfSize = envmapPdf.size();
    float u = uv.x, v = (1 -uv.y);
    int rowId = int(v * pdfSize.y);
    int colId = int(u * pdfSize.x);
    return envmapPdf[make_uint2(colId, rowId ) ];
}

RT_CALLABLE_PROGRAM float4 SampleEnvironmap(float z1, float z2)
{
    int ncols = envcdfH.size().x;
    int nrows = envcdfH.size().y;
    
    // Sample the row 
    float u = 0, v = 0;
    int rowId = 0;
    {
        int left = 0, right = nrows-1;
        while(right > left){
            int mid = (left + right) / 2;
            if(envcdfV[ make_uint2(0, mid) ] >= z1)
                right = mid;
            else if(envcdfV[ make_uint2(0, mid) ] < z1)
                left = mid + 1;
        }
        float up = envcdfV[make_uint2(0, left) ];
        float down = (left == 0) ? 0 : envcdfV[make_uint2(0, left-1) ];
        v = ( (z1 - down) / (up - down) + left) / float(nrows);
        rowId = left;
    }

    // Sample the column
    int colId = 0;
    {
        int left = 0; int right = ncols - 1;
        while(right > left){
            int mid = (left + right) / 2;
            if(envcdfH[ make_uint2(mid, rowId) ] >= z2)
                right = mid;
            else if(envcdfH[ make_uint2(mid, rowId) ] < z2)
                left = mid + 1;
        }
        float up = envcdfH[make_uint2(left, rowId) ];
        float down = (left == 0) ? 0 : envcdfH[make_uint2(left-1, rowId) ];
        u = ((z2 - down) / (up - down) + left) / float(ncols);
        colId = left;
    }

    // Turn uv coordinate into direction
    float3 direc = EnvUVToDirec(u, 1 - v);
    float pdf = envmapPdf[make_uint2(colId, rowId) ];

    float4 direcPdf = make_float4(
                direc, pdf
            );
    return direcPdf;
}
/*********************************************/


/*********************************************/
/**** Function to sample BRDF ****/
/*********************************************/
RT_CALLABLE_PROGRAM float LambDirecToPdf(const float3& L, const float3& N){
    float pdf = dot(L, N) / M_PIf;
    return fmaxf(pdf, 0.0f);
}
RT_CALLABLE_PROGRAM float SpecDirecToPdf(const float3& L, const float3& V, const float3& N, float R){
    float a2 = R * R * R * R;
    float3 H = normalize( (L+V) / 2.0 );
    float NoH = dot(N, H);
    float VoH = dot(V, H);
    float pdf = (a2 * NoH) / (4 * M_PIf * (1 + (a2-1) * NoH)
            *(1 + (a2-1) * NoH) * VoH );
    return fmaxf(pdf, 0.0f);
}
/*********************************************/


// The Camera Model
RT_PROGRAM void pinhole_camera()
{
    size_t2 screen = output_buffer.size();
    
    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x,  0);
    do{
        // Sample pixel using jittering
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed) );
        float2 d = pixel + jitter*jitter_scale;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.radiance = make_float3(0.f);
        prd.countEmitted = false;
        prd.done = false;
        prd.seed = tea<8>(seed, 0);
        prd.depth = 0;
        prd.direction = ray_direction;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            optix::Ray ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon);
            rtTrace(top_object, ray, prd);
 
            prd.depth++;

            if(mode == 9 || mode == 10 || mode == 11){
                if(prd.depth == bounce_num)
                    break;
            }
            
            // Hit the light source or exceed the max depth
            if(prd.done || prd.depth >= max_depth) 
                break;
            
            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }
        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    // Update the output buffer
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);
    if(mode == 0 || mode == 1 || mode == 2 || mode == 9 || mode == 10 || mode == 11){
        pixel_color.x = powf(pixel_color.x, 1.0f/2.2f );
        pixel_color.y = powf(pixel_color.y, 1.0f/2.2f );
        pixel_color.z = powf(pixel_color.z, 1.0f/2.2f );
    }
    if (mode != 6)
        output_buffer[launch_index] = make_color(pixel_color);
    else
        float_buffer[launch_index] = make_float4(pixel_color, 1.0);
}
RT_PROGRAM void exception()
{
    const unsigned int code = rtGetExceptionCode();
    rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
    if(mode != 6)
        output_buffer[launch_index] = make_color( bad_color );
    else
        float_buffer[launch_index] = make_float4(bad_color, 1);
}


RT_PROGRAM void microfacetMIS()
{
    if(mode < 9)
    {
        // Russian roulette termination 
        if(current_prd.depth > rr_begin_depth){
            float pcont = fmaxf(current_prd.attenuation);
            if(rnd(current_prd.seed) >= pcont){
                current_prd.done = true;
                return;
            }
            current_prd.attenuation /= pcont;
        }
    }
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;
    float3 diffuse_color = make_float3(tex2D(albedoMap, texcoord.x, texcoord.y) ) * M_1_PIf;
    float3 normal_detail = make_float3(tex2D(normalMap, texcoord.x, texcoord.y) );
    normal_detail = normalize(2 * normal_detail - 1);
    float roughness  = tex2D(roughMap, texcoord.x, texcoord.y).x;

    // Upate Geometry information
    current_prd.origin = hitpoint;
    unsigned int seed = current_prd.seed;
    float z = rnd(seed);
    seed = tea<8>(seed, 2);
    float z1 = rnd(seed);
    seed = tea<8>(seed, 2);
    float z2 = rnd(seed);
    current_prd.seed = seed;

    normal_detail = normal_detail.x * tangent_direction
        + normal_detail.y * bitangent_direction 
        + normal_detail.z * ffnormal;
    normal_detail = normalize(normal_detail);
    float3 N = normal_detail;
    optix::Onb onb(N);
    float3 V = -ray.direction;

    // Data preparation for the computation
    float3 attenuationPre = current_prd.attenuation;
    float alpha = roughness * roughness;
    float k = (alpha + 2 * roughness + 1) / 8;
    float alpha2 = alpha * alpha;
   
    if(z < 0.5){
        float3 L;
        cosine_sample_hemisphere(z1, z2, L);
        onb.inverse_transform(L);
        current_prd.direction = L;
        current_prd.attenuation = 2 * M_PIf * current_prd.attenuation * diffuse_color;
    }
    else {
        // Compute the half angle 
        float phi = 2 * M_PIf * z1;
        float cosTheta = sqrt( (1 - z2) / (1 + (alpha2 - 1) * z2) );
        float sinTheta = sqrt( 1 - cosTheta * cosTheta);

        float3 H = make_float3(
                sinTheta * cos(phi),
                sinTheta * sin(phi),
                cosTheta);
        onb.inverse_transform(H);
        float3 L = 2 * dot(V, H) * H - V;
        current_prd.direction = L;

        float NoV = fmaxf(dot(N, V), 0.0000001f);
        float NoL = dot(N, L);
        float NoH = fmaxf(dot(N, H), 0.00000001f);
        float VoH = fmaxf(dot(V, H), 0.00000001f);

        if( NoL >= 0){
            float G1 = NoV / (NoV * (1-k) + k);
            float G2 = NoL / (NoL * (1-k) + k);
            float FMi = (-5.55473 * VoH - 6.98316) * VoH;
            float F = F0 + (1 - F0) * pow(2.0f, FMi);
            float reflec = F * G1 * G2 * VoH / NoH / NoV;

            current_prd.attenuation = 2 * current_prd.attenuation * reflec;
        }
        else
            current_prd.attenuation = make_float3(0.0f);
    }

    if(mode != 1 && mode != 10)
    { // Connect with the point light source
        float pLdist = length(plight - hitpoint);
        float3 L = normalize(plight - hitpoint);
        float3 H = normalize((L + V) / 2.0f );
        float NoL = dot(N, L);
        float NoV = fmaxf(dot(N, V), 1e-6);
        float NoH = fmaxf(dot(N, H), 1e-6);
        float VoH = fmaxf(dot(V, H), 1e-6);

        // Connect with the point light source and compute the direct contribution 
        if (NoL > 0.0f){
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            Ray shadow_ray = make_Ray(hitpoint, L, shadow_ray_type, scene_epsilon, pLdist-scene_epsilon);
            rtTrace(top_object, shadow_ray, shadow_prd);
            if(!shadow_prd.inShadow){
                float FMi = (-5.55473 * VoH - 6.98316) * VoH;
                float frac0 = F0 + (1 - F0) * pow(2.0f, FMi);
                float frac = frac0 * alpha2;
                float nom0 = NoH * NoH * (alpha2 - 1) + 1;
                float nom1 = NoV * (1 - k) + k;
                float nom2 = NoL * (1 - k) + k;
                float nom = fmaxf(4 * M_PIf * nom0 * nom0 * nom1 * nom2, 1e-6);
                float spec = frac / nom;

                if(mode >= 9){
                    if(current_prd.depth == bounce_num-1){
                        current_prd.result += attenuationPre * (diffuse_color + spec) * 
                            NoL * lightIntensity / pLdist / pLdist;
                    }
                }
                else{
                    current_prd.result += attenuationPre * (diffuse_color + spec) * 
                        NoL * lightIntensity / pLdist / pLdist;
                }
            }
        }
    }
    

    if(mode != 0 && mode != 9)
    { 
        unsigned int seed = current_prd.seed;
        seed = tea<8>(seed, 2);
        float z3 = rnd(seed);
        seed = tea<8>(seed, 2);
        float z4 = rnd(seed);
        seed = tea<8>(seed, 2);
        float z5 = rnd(seed);
        current_prd.seed = seed;

        // Sample According to Environment Map
        {
            // Connect with Environment Map
            float4 directPdf = SampleEnvironmap(z3, z4);
            float3 L = normalize( make_float3(
                        directPdf.x, directPdf.y, directPdf.z) );
            float pdf = fmaxf(directPdf.w, 1e-6 );
            float3 H = normalize((L + V) / 2.0f );
            float NoL = dot(N, L);
            float NoV = fmaxf(dot(N, V), 1e-6);
            float NoH = fmaxf(dot(N, H), 1e-6);
            float VoH = fmaxf(dot(V, H), 1e-6);
     
            // Get the radiance of the environment map
            float2 uv = EnvDirecToUV(L);
            float3 radiance =  make_float3( tex2D(envmap, uv.x, uv.y) );
         
            // Connect with the environment and compute the direct contribution 
            if (NoL > 0.0f){
                PerRayData_pathtrace_shadow shadow_prd;
                shadow_prd.inShadow = false;
                Ray shadow_ray = make_Ray(hitpoint, L, shadow_ray_type, scene_epsilon, infiniteFar-scene_epsilon);
                rtTrace(top_object, shadow_ray, shadow_prd);
                if(!shadow_prd.inShadow){
                    float FMi = (-5.55473 * VoH - 6.98316) * VoH;
                    float frac0 = F0 + (1 - F0) * pow(2.0f, FMi);
                    float frac = frac0 * alpha2;
                    float nom0 = NoH * NoH * (alpha2 - 1) + 1;
                    float nom1 = NoV * (1 - k) + k;
                    float nom2 = NoL * (1 - k) + k;
                    float nom = fmaxf(4 * M_PIf * nom0 * nom0 * nom1 * nom2, 1e-6);
                    float spec = frac / nom;
         
                    float Lambpdf = LambDirecToPdf(L, N);
                    float Specpdf = SpecDirecToPdf(L, V, N, roughness);
                    float Envpdf = EnvDirecToPdf(L);
                    float W = Envpdf * Envpdf / (Lambpdf*Lambpdf 
                            + Envpdf*Envpdf + Specpdf*Specpdf); 
                    if(mode >= 9){
                        if(current_prd.depth == bounce_num - 1){
                            current_prd.result += W * attenuationPre * (diffuse_color + spec) * 
                                NoL * radiance / pdf; 
                        }
                    }
                    else{
                        current_prd.result += W * attenuationPre * (diffuse_color + spec) * 
                            NoL * radiance / pdf; 
                    }
                }
            }
        }

        // Sampling According to BRDF
        {
            if(z3 < 0.5){
                float3 L;
                cosine_sample_hemisphere(z4, z5, L);
                onb.inverse_transform(L);
                PerRayData_pathtrace_shadow shadow_prd;
                shadow_prd.inShadow = false;
                Ray shadow_ray = make_Ray(hitpoint, L, shadow_ray_type, scene_epsilon, infiniteFar-scene_epsilon);
                rtTrace(top_object, shadow_ray, shadow_prd);
                if( !shadow_prd.inShadow ){
                    float2 uv = EnvDirecToUV(L);
                    float3 radiance =  make_float3( tex2D(envmap, uv.x, uv.y) );
                    
                    float Lambpdf = LambDirecToPdf(L, N);
                    float Specpdf = SpecDirecToPdf(L, V, N, roughness);
                    float Envpdf = EnvDirecToPdf(L);
                    float W = 2 * (Lambpdf * Lambpdf + Specpdf * Specpdf) / (Lambpdf*Lambpdf 
                            + Envpdf*Envpdf + Specpdf*Specpdf);        
                    if(mode >= 9){
                        if(current_prd.depth == bounce_num - 1){
                            current_prd.result += W * radiance * M_PIf * attenuationPre * diffuse_color;
                        }
                    }
                    else{
                        current_prd.result += W * radiance * M_PIf * attenuationPre * diffuse_color;
                    }
                }
            }
            else {
                // Compute the half angle 
                float phi = 2 * M_PIf * z4;
                float cosTheta = sqrt( (1 - z5) / (1 + (alpha2 - 1) * z5) );
                float sinTheta = sqrt( 1 - cosTheta * cosTheta);
  
                float3 H = make_float3(
                        sinTheta * cos(phi),
                        sinTheta * sin(phi),
                        cosTheta);
                onb.inverse_transform(H);
                float3 L = 2 * dot(V, H) * H - V;
  
                float NoV = fmaxf(dot(N, V), 0.0000001f);
                float NoL = dot(N, L);
                float NoH = fmaxf(dot(N, H), 0.00000001f);
                float VoH = fmaxf(dot(V, H), 0.00000001f);
  
                if( NoL >= 0){
                    PerRayData_pathtrace_shadow shadow_prd;
                    shadow_prd.inShadow = false;
                    Ray shadow_ray = make_Ray(hitpoint, L, shadow_ray_type, scene_epsilon, infiniteFar-scene_epsilon);
                    rtTrace(top_object, shadow_ray, shadow_prd);
                    if(!shadow_prd.inShadow){
                        float G1 = NoV / (NoV * (1-k) + k);
                        float G2 = NoL / (NoL * (1-k) + k);
                        float FMi = (-5.55473 * VoH - 6.98316) * VoH;
                        float F = F0 + (1 - F0) * pow(2.0f, FMi);
                        float reflec = F * G1 * G2 * VoH / NoH / NoV;

                        float2 uv = EnvDirecToUV(L);
                        float3 radiance =  make_float3( tex2D(envmap, uv.x, uv.y) );
                        float Lambpdf = LambDirecToPdf(L, N);
                        float Specpdf = SpecDirecToPdf(L, V, N, roughness);
                        float Envpdf = EnvDirecToPdf(L);
                        float W = 2 * (Lambpdf * Lambpdf + Specpdf * Specpdf) / (Lambpdf*Lambpdf 
                                + Envpdf*Envpdf + Specpdf*Specpdf);        
                        if(mode >= 9){
                            if(current_prd.depth == bounce_num - 1){
                                current_prd.result +=  W * attenuationPre * radiance * reflec;
                            }
                        }
                        else{
                            current_prd.result +=  W * attenuationPre * radiance * reflec;
                        }
                    }
                }
            }
        }
    }
}

// Rendering BRDF parameter
RT_PROGRAM void albedo()
{
    float3 diffuse_color = make_float3(tex2D(albedoMap, texcoord.x, texcoord.y) );
    current_prd.result = diffuse_color;
}
RT_PROGRAM void roughness()
{
    float3 roughness = make_float3(tex2D(roughMap, texcoord.x, texcoord.y) );
    current_prd.result = roughness;
}
RT_PROGRAM void segmentation()
{
    current_prd.result = make_float3(1.0);
}
RT_PROGRAM void depth()
{
    float3 Z = normalize(W);
    current_prd.result = t_hit * make_float3(dot(Z, ray.direction) );
}
RT_PROGRAM void normal()
{
    float3 world_shading_normal   = normalize( rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal) );
    float3 world_geometric_normal = normalize( rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal) );
    float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal );

    float3 normal_detail = make_float3(tex2D(normalMap, texcoord.x, texcoord.y) );
    normal_detail = normalize(2*normal_detail - 1);
    normal_detail = normal_detail.x * tangent_direction
        + normal_detail.y * bitangent_direction 
        + normal_detail.z * ffnormal;
    normal_detail = normalize(normal_detail);

    float3 X = normalize(U);
    float3 Y = normalize(V);
    float3 Z = -normalize(W);
    float xComp = 0.5 * dot(X, normal_detail) + 0.5;
    float yComp = 0.5 * dot(Y, normal_detail) + 0.5;
    float zComp = 0.5 * dot(Z, normal_detail) + 0.5;
    xComp = fminf(fmaxf(xComp, 0.0f), 1.0f);
    yComp = fminf(fmaxf(yComp, 0.0f), 1.0f);
    zComp = fminf(fmaxf(zComp, 0.0f), 1.0f);

    current_prd.result = make_float3(xComp, yComp, zComp);
}
RT_PROGRAM void normalCoarse()
{
    float3 world_shading_normal   = normalize( rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal) );
    float3 world_geometric_normal = normalize( rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal) );
    float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal );

    float3 X = normalize(U);
    float3 Y = normalize(V);
    float3 Z = -normalize(W);
    float xComp = 0.5 * dot(X, ffnormal) + 0.5;
    float yComp = 0.5 * dot(Y, ffnormal) + 0.5;
    float zComp = 0.5 * dot(Z, ffnormal) + 0.5;
    xComp = fminf(fmaxf(xComp, 0.0f), 1.0f);
    yComp = fminf(fmaxf(yComp, 0.0f), 1.0f);
    zComp = fminf(fmaxf(zComp, 0.0f), 1.0f);
    current_prd.result = make_float3(xComp, yComp, zComp);
}
RT_PROGRAM void uvmap()
{
    current_prd.result = make_float3(texcoord.x, texcoord.y, 1.0);
}


RT_PROGRAM void envmap_miss() {
    float2 uv = EnvDirecToUV(current_prd.direction);
    if(current_prd.depth == 0){
        if(mode != 0 && mode != 9)
            current_prd.result =  make_float3( tex2D(envmap, uv.x, uv.y) );
        else 
            current_prd.result = make_float3(0.0f);
    }
    current_prd.done = true;
}

RT_PROGRAM void miss(){
    current_prd.radiance = make_float3(0.0f);
    current_prd.done = true;
}

// -------------------------------------------------------------
//  Shadow any-hit
// -------------------------------------------------------------
rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );
RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}

