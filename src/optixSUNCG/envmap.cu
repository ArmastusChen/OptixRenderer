#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "commonStructs.h"
#include "areaLight.h"
#include <vector>

using namespace optix;


rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Environmental Lighting 
rtDeclareVariable(int, isEnvmap, , );
rtTextureSampler<float4, 2> envmap;
rtTextureSampler<float4, 2> envmapDirec;
rtBuffer<float, 2> envcdfV;
rtBuffer<float, 2> envcdfH;
rtBuffer<float, 2> envpdf;
rtDeclareVariable(float, infiniteFar, , );


RT_CALLABLE_PROGRAM float3 EnvUVToDirec(float u, float v){ 
    // Turn uv coordinate into direction
    float theta = 2 * (u - 0.5) * M_PIf;
    float phi = M_PIf * (1 - v); 
    return make_float3(
                sinf(phi) * sinf(theta),
                cosf(phi),
                sinf(phi) * cosf(theta)
            );
}
RT_CALLABLE_PROGRAM float2 EnvDirecToUV(const float3& direc){ 
    float theta = atan2f( direc.x, direc.z );
    float phi = M_PIf - acosf(direc.y );
    float u = theta * (0.5f * M_1_PIf) + 0.5;
    if(u > 1)
        u = u-1;
    float v     = phi / M_PIf;
    return make_float2(u, v);
}
RT_CALLABLE_PROGRAM float EnvDirecToPdf(const float3& direc){
    float2 uv = EnvDirecToUV(direc);
    size_t2 pdfSize = envpdf.size();
    float u = uv.x, v = uv.y;
    int rowId = int(v * (pdfSize.y - 1) );
    int colId = int(u * (pdfSize.x - 1) );
    return envpdf[make_uint2(colId, rowId ) ];
}
RT_PROGRAM void envmap_miss(){
    if(isEnvmap == 0){
        prd_radiance.attenuation = make_float3(0.0);
    }
    else if(isEnvmap == 1){    
        float2 uv = EnvDirecToUV(prd_radiance.direction);

        if(prd_radiance.depth == 0){
            prd_radiance.radiance = make_float3(tex2D(envmapDirec, uv.x, uv.y) ); 
        }
        else{
            float3 radiance = make_float3(tex2D(envmap, uv.x, uv.y) );
            // Multiple Importance Sampling 
            float pdfSolidEnv = EnvDirecToPdf(prd_radiance.direction);
            float pdfSolidBRDF = prd_radiance.pdf;
            float pdfSolidEnv2 = pdfSolidEnv * pdfSolidEnv;
            float pdfSolidBRDF2 = pdfSolidBRDF * pdfSolidBRDF;

            prd_radiance.radiance += radiance  * pdfSolidBRDF2 / fmaxf(pdfSolidBRDF2 + pdfSolidEnv2, 1e-6)* prd_radiance.attenuation;
        }
    }
    prd_radiance.done = true;
}

RT_PROGRAM void miss(){
    prd_radiance.radiance = make_float3(0.0);
    prd_radiance.done = true;
}
