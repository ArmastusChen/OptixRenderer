#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "commonStructs.h"

using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

rtDeclareVariable(float3, radiance, , );
rtDeclareVariable(float, areaSum, , );


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    if(prd_radiance.depth == 0){
        // Directly hit the light
        prd_radiance.radiance = radiance;
    }
    else{
        // Use MIS to compute the radiance
        float3 hitPoint = ray.origin + t_hit * ray.direction;
        float Dist = length(hitPoint - prd_radiance.origin);
        float3 L = normalize(hitPoint - prd_radiance.origin);
        float cosPhi = dot(L, ffnormal);
        if (cosPhi < 0) cosPhi = -cosPhi;
        if (cosPhi < 1e-6) cosPhi = 0;
        
        float pdfAreaBRDF = prd_radiance.pdf * cosPhi / Dist / Dist;
        float pdfAreaLight = 1 / areaSum;

        float pdfAreaBRDF2 = pdfAreaBRDF * pdfAreaBRDF;
        float pdfAreaLight2 = pdfAreaLight * pdfAreaLight;
       
        prd_radiance.radiance += radiance * pdfAreaBRDF2 / (pdfAreaBRDF2 + pdfAreaLight2) * prd_radiance.attenuation;
    }
    prd_radiance.done = true;
}


RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}
