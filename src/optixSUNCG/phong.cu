#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "commonStructs.h"
#include "sutil/lightStructs.h"
#include <vector>

using namespace optix;


rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float, scene_epsilon, , );

// Materials
rtDeclareVariable( float3, albedo, , );
rtTextureSampler<float4, 2> albedoMap;
rtDeclareVariable( int, isAlbedoTexture, , );
rtDeclareVariable( float3, specular, , );
rtTextureSampler<float4, 2> specularMap;
rtDeclareVariable( int, isSpecularTexture, , );
rtDeclareVariable( float, glossy, , );
rtTextureSampler<float4, 2> glossyMap;
rtDeclareVariable( int, isGlossyTexture, , );
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable(int, isNormalTexture, , );
rtDeclareVariable(float, F0, , );

// Area Light Buffer
rtDeclareVariable(int, isAreaLight, , );
rtDeclareVariable(int, areaTriangleNum, , );
rtBuffer<areaLight> areaLights;
rtBuffer<float> areaLightCDF;
rtBuffer<float> areaLightPDF;

// Environmental Lighting 
rtDeclareVariable(int, isEnvmap, , );
rtTextureSampler<float4, 2> envmap;
rtBuffer<float, 2> envcdfV;
rtBuffer<float, 2> envcdfH;
rtBuffer<float, 2> envpdf;
rtDeclareVariable(float, infiniteFar, , );

// Point lighting 
rtDeclareVariable(int, isPointLight, , );
rtDeclareVariable(int, pointLightNum, , );
rtBuffer<Point> pointLights;


// Geometry Group
rtDeclareVariable( rtObject, top_object, , );


// Samplling the area Light
RT_CALLABLE_PROGRAM void sampleAreaLight(unsigned int& seed, float3& radiance, float3& position, float3& normal, float& pdfAreaLight){
    float randf = rnd(seed);

    int left = 0, right = areaTriangleNum;
    int middle = int( (left + right) / 2);
    while(left < right){
        if(areaLightCDF[middle] <= randf)
            left = middle + 1;
        else if(areaLightCDF[middle] > randf)
            right = middle;
        middle = int( (left + right) / 2);
    }
    areaLight L = areaLights[left];
    
    float3 v1 = L.vertices[0];
    float3 v2 = L.vertices[1];
    float3 v3 = L.vertices[2];

    normal = cross(v2 - v1, v3 - v1);
    float area = 0.5 * length(normal);
    normal = normalize(normal);

    float ep1 = rnd(seed);
    float ep2 = rnd(seed);
    
    float u = 1 - sqrt(ep1);
    float v = ep2 * sqrt(ep1);

    position = v1 + (v2 - v1) * u + (v3 - v1) * v;

    radiance = L.radiance;
    pdfAreaLight = areaLightPDF[left] /  fmaxf(area, 1e-14);
}

// Sampling the environmnetal light
RT_CALLABLE_PROGRAM float3 EnvUVToDirec(float u, float v){ 
    // Turn uv coordinate into direction
    float theta = 2 * (u - 0.5) * M_PIf;
    float phi = M_PIf * (1 - v); 
    return make_float3(
                sinf(phi) * sinf(theta),
                cosf(phi),
                sinf(phi) * cosf(theta)
            );
}
RT_CALLABLE_PROGRAM float2 EnvDirecToUV(const float3& direc){ 
    float theta = atan2f( direc.x, direc.z );
    float phi = M_PIf - acosf(direc.y );
    float u = theta * (0.5f * M_1_PIf) + 0.5;
    if(u > 1)
        u = u-1;
    float v     = phi / M_PIf;
    return make_float2(u, v);
}
RT_CALLABLE_PROGRAM float EnvDirecToPdf(const float3& direc){
    float2 uv = EnvDirecToUV(direc);
    size_t2 pdfSize = envpdf.size();
    float u = uv.x, v = uv.y;
    int rowId = int(v * (pdfSize.y-1) );
    int colId = int(u * (pdfSize.x-1) );
    return envpdf[make_uint2(colId, rowId ) ];
}
RT_CALLABLE_PROGRAM void sampleEnvironmapLight(unsigned int& seed, float3& radiance, float3& direction, float& pdfSolidEnv){
    float z1 = rnd(seed);
    float z2 = rnd(seed);
    
    int ncols = envcdfH.size().x;
    int nrows = envcdfH.size().y;

    // Sample the row 
    float u = 0, v = 0;
    int rowId = 0;
    {
        int left = 0, right = nrows-1;
        while(right > left){
            int mid = (left + right) / 2;
            if(envcdfV[ make_uint2(0, mid) ] >= z1)
                right = mid;
            else if(envcdfV[ make_uint2(0, mid) ] < z1)
                left = mid + 1;
        }
        float up = envcdfV[make_uint2(0, left) ];
        float down = (left == 0) ? 0 : envcdfV[make_uint2(0, left-1) ];
        v = ( (z1 - down) / (up - down) + left) / float(nrows);
        rowId = left;
    }

    // Sample the column
    int colId = 0;
    {
        int left = 0; int right = ncols - 1;
        while(right > left){
            int mid = (left + right) / 2;
            if(envcdfH[ make_uint2(mid, rowId) ] >= z2)
                right = mid;
            else if(envcdfH[ make_uint2(mid, rowId) ] < z2)
                left = mid + 1;
        }
        float up = envcdfH[make_uint2(left, rowId) ];
        float down = (left == 0) ? 0 : envcdfH[make_uint2(left-1, rowId) ];
        u = ((z2 - down) / (up - down) + left) / float(ncols);
        colId = left;
    }
    
    // Turn uv coordinate into direction
    direction = EnvUVToDirec(u, v);
    pdfSolidEnv = envpdf[make_uint2(colId, rowId) ];
    radiance = make_float3(tex2D(envmap, u, v) );
}


// Computing the pdfSolidAngle of BRDF giving a direction 
RT_CALLABLE_PROGRAM float LambertianPdf(const float3& L, const float3& N)
{
    float NoL = fmaxf(dot(N, L), 1e-14);
    float pdf = NoL / M_PIf;
    return fmaxf(pdf, 1e-14f);
}
RT_CALLABLE_PROGRAM float SpecularPdf(const float3& L, const float3& N, const float3& R, 
        float glossyValue)
{ 
    float RoL = dot(R, L);
    if(RoL < 1e-14) RoL = 0;
    float pdf = (glossyValue + 2) / (2*M_PIf) * pow(RoL, fmaxf(glossyValue, 1e-14) );
    return fmaxf(pdf, 1e-14);
}
RT_CALLABLE_PROGRAM float pdf(const float3& L, const float3& N, const float3& R, const float3& albedoValue, const float3& specularValue, float glossyValue)
{
    float pdfLambertian = LambertianPdf(L, N);
    float pdfSpecular = SpecularPdf(L, N, R, glossyValue);
    float albedoStr = length(albedoValue );
    float specularStr = length(specularValue );
    float pdf = (albedoStr * pdfLambertian + specularStr * pdfLambertian) / fmaxf(albedoStr + specularStr, 1e-14);
    return fmaxf(pdf, 1e-14);
}

RT_CALLABLE_PROGRAM float3 evaluate(const float3& albedoValue, const float3& specularValue, const float3& N, const float glossyValue, 
        const float3& L, const float3& R, const float3& radiance)
{
    float NoL = fmaxf(dot(N, L), 1e-14);

    float RoL = dot(R, L);
    if(RoL < 1e-14) RoL = 0;

    float3 lambertianTerm = albedoValue / M_PIf;
    float3 specularTerm = specularValue / (2*M_PIf) * (glossyValue + 2) * pow(RoL, fmaxf(glossyValue, 1e-14) );
    return (lambertianTerm + specularTerm) * radiance * NoL;
}

RT_CALLABLE_PROGRAM void sample(unsigned& seed, 
        const float3& albedoValue, const float3& specularValue, const float3& N, const float glossyValue, const float3& R, 
        optix::Onb onb, 
        float3& attenuation, float3& direction, float& pdfSolid)
{
    const float z1 = rnd( seed );
    const float z2 = rnd( seed );
    const float z = rnd( seed );
    
    float albedoStr = length(albedoValue );
    float specularStr = length(specularValue );

    float3 L;
    if(z <= albedoStr / fmaxf(albedoStr + specularStr, 1e-14) || (albedoStr + specularStr) < 1e-14 ){
        cosine_sample_hemisphere(z1, z2, L);
        onb.inverse_transform(L);
        attenuation = attenuation * albedoValue * (albedoStr + specularStr) / fmaxf(albedoStr, 1e-14);
    }
    else{
        float z1_1_nP1 = pow(z1, 1 / (glossyValue +1) );
        float z1_2_nP1 = z1_1_nP1 * z1_1_nP1;
        L = make_float3(
                sqrt(1 - z1_2_nP1) * cos(2 * M_PIf * z2), 
                sqrt(1 - z1_2_nP1) * sin(2 * M_PIf * z2),
                z1_1_nP1
                );
        optix::Onb ronb(R);
        ronb.inverse_transform(L);
        float NoL = fmaxf(dot(N, L), 1e-14);
        attenuation = attenuation * specularValue * NoL * (albedoStr + specularStr) / fmaxf(specularStr, 1e-14); 
    }
    direction = L;
    pdfSolid = pdf(L, N, R, albedoValue, specularValue, glossyValue);
    return;
}


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
 
    float3 albedoValue;
    if(isAlbedoTexture == 0){
        albedoValue = albedo;
    }
    else{
        albedoValue = make_float3(tex2D(albedoMap, texcoord.x, texcoord.y) );
        albedoValue.x = pow(albedoValue.x, 2.2);
        albedoValue.y = pow(albedoValue.y, 2.2);
        albedoValue.z = pow(albedoValue.z, 2.2);
    }

    float3 specularValue;
    if(isSpecularTexture == 0){
        specularValue = specular;
    }
    else{
        specularValue = make_float3(tex2D(specularMap, texcoord.x, texcoord.y) );
        specularValue.x = pow(specularValue.x, 2.2);
        specularValue.y = pow(specularValue.y, 2.2);
        specularValue.z = pow(specularValue.z, 2.2);
    }



    float3 colorSum = fmaxf(albedoValue + specularValue, make_float3(1e-14f) );
    float colorMax= colorSum.x;
    if(colorMax < colorSum.y) colorMax = colorSum.y;
    if(colorMax < colorSum.z) colorMax = colorSum.z;
    colorMax = fmaxf(colorMax, 1e-14);

    if(colorMax > 1){
        specularValue = specularValue / colorMax;
        albedoValue = albedoValue / colorMax;
    }

    float glossyValue = (isGlossyTexture == 0) ? glossy :
        tex2D(glossyMap, texcoord.x, texcoord.y).x;
    
    float3 V = normalize(-ray.direction );    
    if(dot(ffnormal, V) < 0)
        ffnormal = -ffnormal;
    
    float3 N;
    if( isNormalTexture == 0){
        N = ffnormal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x, texcoord.y) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * ffnormal;
    }
    N = normalize(N );
    optix::Onb onb(N );
    
    float3 hitPoint = ray.origin + t_hit * ray.direction;
    prd_radiance.origin = hitPoint;

    float3 R = 2 * dot(V, N) * N - V;

    // Connect to the area Light
    {
        if(isAreaLight == 1){
            float3 position, radiance, normal;
            float pdfAreaLight;
            sampleAreaLight(prd_radiance.seed, radiance, position, normal, pdfAreaLight);
   
            float Dist = length(position - hitPoint);
            float3 L = normalize(position - hitPoint);

            if(fmaxf(dot(N, L), 0.0) * fmaxf(dot(N, V), 0.0) > 0 ){
                float cosPhi = dot(L, normal);
                cosPhi = (cosPhi < 0) ? -cosPhi : cosPhi;

                Ray shadowRay = make_Ray(hitPoint, L, 1, scene_epsilon, Dist - scene_epsilon);
                PerRayData_shadow prd_shadow; 
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, specularValue, N, glossyValue, L, R, radiance) * cosPhi / Dist / Dist;
                    float pdfSolidBRDF = pdf(L, N, R, albedoValue, specularValue, glossyValue);
                    float pdfAreaBRDF = pdfSolidBRDF * cosPhi / Dist / Dist;

                    float pdfAreaLight2 = pdfAreaLight * pdfAreaLight;
                    float pdfAreaBRDF2 = pdfAreaBRDF * pdfAreaBRDF;

                    prd_radiance.radiance += intensity * pdfAreaLight / (pdfAreaBRDF2 + pdfAreaLight2) * prd_radiance.attenuation;
                }
            }
        }
    }
    
    // Connect to point light 
    {
        if(isPointLight == 1){
            // Connect to every point light 
            for(int i = 0; i < pointLightNum; i++){
                float3 position = pointLights[i].position;
                float3 radiance = pointLights[i].intensity;
                float3 L = normalize(position - hitPoint);
                float Dist = length(position - hitPoint);

                if(fmaxf(dot(N, L), 0.0) * fmaxf(dot(N, V), 0.0) > 0 ){
                    Ray shadowRay = make_Ray(hitPoint + 0.1 * L * scene_epsilon, L, 1, scene_epsilon, Dist - scene_epsilon);
                    PerRayData_shadow prd_shadow; 
                    prd_shadow.inShadow = false;
                    rtTrace(top_object, shadowRay, prd_shadow);
                    if(prd_shadow.inShadow == false)
                    {
                        float3 intensity = evaluate(albedoValue, specularValue, N, glossyValue, L, R, radiance) / Dist/ Dist;
                        prd_radiance.radiance += intensity * prd_radiance.attenuation;
                    }
                }
            }
        }
    }

    // Connect to the environmental map 
    { 
        if(isEnvmap == 1){
            float3 L, radiance;
            float pdfSolidEnv;
            sampleEnvironmapLight(prd_radiance.seed, radiance, L, pdfSolidEnv);

            if( fmaxf(dot(L, N), 0.0) * fmaxf(dot(V, N), 0.0) > 0){
                Ray shadowRay = make_Ray(hitPoint + 0.1 * scene_epsilon, L, 1, scene_epsilon, infiniteFar);
                PerRayData_shadow prd_shadow;
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, specularValue, N, glossyValue, L, R, radiance);
                    float pdfSolidBRDF = pdf(L, N, R, albedoValue, specularValue, glossyValue);
                    float pdfSolidBRDF2 = pdfSolidBRDF * pdfSolidBRDF;
                    float pdfSolidEnv2 = pdfSolidEnv * pdfSolidEnv;
                    prd_radiance.radiance += intensity * pdfSolidEnv / (pdfSolidEnv2 + pdfSolidBRDF2) * prd_radiance.attenuation; 
                }
            }
        }
    }

    // Sammple the new ray 
    sample(prd_radiance.seed, 
        albedoValue, specularValue, N, glossyValue, R, 
        onb, 
        prd_radiance.attenuation, prd_radiance.direction, prd_radiance.pdf);

}    

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

