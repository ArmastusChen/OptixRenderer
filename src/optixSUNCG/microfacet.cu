#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "commonStructs.h"
#include "areaLight.h"
#include "point.h"
#include <vector>

using namespace optix;


rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float, scene_epsilon, , );

// Materials
rtDeclareVariable( float3, albedo, , );
rtTextureSampler<float4, 2> albedoMap;
rtDeclareVariable( int, isAlbedoTexture, , );
rtDeclareVariable( float, rough, , );
rtTextureSampler<float4, 2> roughMap;
rtDeclareVariable( int, isRoughTexture, , );
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable(int, isNormalTexture, , );
rtDeclareVariable(float, F0, , );
rtDeclareVariable( float, metallic, , );
rtDeclareVariable( int, isMetallicTexture, ,  );
rtTextureSampler<float4, 2> metallicMap;

// Area Light Buffer
rtDeclareVariable(int, isAreaLight, , );
rtDeclareVariable(int, areaTriangleNum, , );
rtBuffer<areaLight> areaLights;
rtBuffer<float> areaLightCDF;
rtBuffer<float> areaLightPDF;

// Environmental Lighting 
rtDeclareVariable(int, isEnvmap, , );
rtTextureSampler<float4, 2> envmap;
rtBuffer<float, 2> envcdfV;
rtBuffer<float, 2> envcdfH;
rtBuffer<float, 2> envpdf;
rtDeclareVariable(float, infiniteFar, , );

// Point lighting 
rtDeclareVariable(int, isPointLight, , );
rtDeclareVariable(int, pointLightNum, , );
rtBuffer<Point> pointLights;


// Geometry Group
rtDeclareVariable( rtObject, top_object, , );


// Samplling the area Light
RT_CALLABLE_PROGRAM void sampleAreaLight(unsigned int& seed, float3& radiance, float3& position, float3& normal, float& pdfAreaLight){
    float randf = rnd(seed);

    int left = 0, right = areaTriangleNum;
    int middle = int( (left + right) / 2);
    while(left < right){
        if(areaLightCDF[middle] <= randf)
            left = middle + 1;
        else if(areaLightCDF[middle] > randf)
            right = middle;
        middle = int( (left + right) / 2);
    }
    areaLight L = areaLights[left];
    
    float3 v1 = L.vertices[0];
    float3 v2 = L.vertices[1];
    float3 v3 = L.vertices[2];

    normal = cross(v2 - v1, v3 - v1);
    float area = 0.5 * length(normal );
    normal = normalize(normal);

    float ep1 = rnd(seed);
    float ep2 = rnd(seed);
    
    float u = 1 - sqrt(ep1);
    float v = ep2 * sqrt(ep1);

    position = v1 + (v2 - v1) * u + (v3 - v1) * v;

    radiance = L.radiance;
    pdfAreaLight = areaLightPDF[left] /  fmaxf(area, 1e-10);
}

// Sampling the environmnetal light
RT_CALLABLE_PROGRAM float3 EnvUVToDirec(float u, float v){ 
    // Turn uv coordinate into direction
    float theta = 2 * (u - 0.5) * M_PIf;
    float phi = M_PIf * (1 - v); 
    return make_float3(
                sinf(phi) * sinf(theta),
                cosf(phi),
                sinf(phi) * cosf(theta)
            );
}
RT_CALLABLE_PROGRAM float2 EnvDirecToUV(const float3& direc){ 
    float theta = atan2f( direc.x, direc.z );
    float phi = M_PIf - acosf(direc.y );
    float u = theta * (0.5f * M_1_PIf) + 0.5;
    if(u > 1)
        u = u-1;
    float v     = phi / M_PIf;
    return make_float2(u, v);
}
RT_CALLABLE_PROGRAM float EnvDirecToPdf(const float3& direc){
    float2 uv = EnvDirecToUV(direc);
    size_t2 pdfSize = envpdf.size();
    float u = uv.x, v = uv.y;
    int rowId = int(v * pdfSize.y);
    int colId = int(u * pdfSize.x);
    return envpdf[make_uint2(colId, rowId ) ];
}
RT_CALLABLE_PROGRAM void sampleEnvironmapLight(unsigned int& seed, float3& radiance, float3& direction, float& pdfSolidEnv){
    float z1 = rnd(seed);
    float z2 = rnd(seed);
    
    int ncols = envcdfH.size().x;
    int nrows = envcdfH.size().y;

    // Sample the row 
    float u = 0, v = 0;
    int rowId = 0;
    {
        int left = 0, right = nrows-1;
        while(right > left){
            int mid = (left + right) / 2;
            if(envcdfV[ make_uint2(0, mid) ] >= z1)
                right = mid;
            else if(envcdfV[ make_uint2(0, mid) ] < z1)
                left = mid + 1;
        }
        float up = envcdfV[make_uint2(0, left) ];
        float down = (left == 0) ? 0 : envcdfV[make_uint2(0, left-1) ];
        v = ( (z1 - down) / (up - down) + left) / float(nrows);
        rowId = left;
    }

    // Sample the column
    int colId = 0;
    {
        int left = 0; int right = ncols - 1;
        while(right > left){
            int mid = (left + right) / 2;
            if(envcdfH[ make_uint2(mid, rowId) ] >= z2)
                right = mid;
            else if(envcdfH[ make_uint2(mid, rowId) ] < z2)
                left = mid + 1;
        }
        float up = envcdfH[make_uint2(left, rowId) ];
        float down = (left == 0) ? 0 : envcdfH[make_uint2(left-1, rowId) ];
        u = ((z2 - down) / (up - down) + left) / float(ncols);
        colId = left;
    }
    
    // Turn uv coordinate into direction
    direction = EnvUVToDirec(u, v);
    pdfSolidEnv = envpdf[make_uint2(colId, rowId) ];
    radiance = make_float3(tex2D(envmap, u, v) );
}


// Computing the pdfSolidAngle of BRDF giving a direction 
RT_CALLABLE_PROGRAM float LambertianPdf(const float3& L, const float3& N)
{
    float NoL = dot(N, L);
    if(NoL < 0) NoL = -NoL;
    if(NoL < 1e-6) NoL = 0;
    float pdf = NoL / M_PIf;
    return fmaxf(pdf, 1e-6f);
}
RT_CALLABLE_PROGRAM float SpecularPdf(const float3& L, const float3& V, const float3& N, float R)
{
    float a2 = R * R * R * R;
    float3 H = normalize( (L+V) / 2.0 );
    float NoH = dot(N, H);
    float VoH = dot(V, H);
    float pdf = (a2 * NoH) / fmaxf( (4 * M_PIf * (1 + (a2-1) * NoH)
            *(1 + (a2-1) * NoH) * VoH ), 1e-6);
    return fmaxf(pdf, 1e-6f);
}
RT_CALLABLE_PROGRAM float pdf(const float3& L, const float3& V, const float3& N, float R)
{
    float pdfLambertian = LambertianPdf(L, N);
    float pdfSpecular = SpecularPdf(L, V, N, R);
    return pdfLambertian * 0.5 + pdfSpecular * 0.5;
}

RT_CALLABLE_PROGRAM float3 evaluate(const float3& albedoValue, const float3& N, const float rough, const float3& fresnel, 
        const float3& V, const float3& L, const float3& radiance)
{
    float alpha = rough * rough;
    float k = (alpha + 2 * rough + 1) / 8.0;
    float alpha2 = alpha * alpha;
    
    float3 H = normalize((L + V) / 2.0f );
    float NoL = dot(N, L);
    float NoV = fmaxf(dot(N, V), 1e-12);
    float NoH = fmaxf(dot(N, H), 1e-12);
    float VoH = fmaxf(dot(V, H), 1e-12);

    float FMi = (-5.55473 * VoH - 6.98316) * VoH;
    float3 frac0 = fresnel + (1 - fresnel) * pow(2.0f, FMi);
    float3 frac = frac0 * alpha2;
    float nom0 = NoH * NoH * (alpha2 - 1) + 1;
    float nom1 = NoV * (1 - k) + k;
    float nom2 = NoL * (1 - k) + k;
    float nom = fmaxf(4 * M_PIf * nom0 * nom0 * nom1 * nom2, 1e-12);
    float3 spec = frac / nom;
         
    float3 intensity = (albedoValue / M_PIf + spec) * NoL * radiance; 
    return intensity;
}

RT_CALLABLE_PROGRAM void sample(unsigned& seed, 
        const float3& albedoValue, const float3& N, const float rough, const float3& fresnel, const float3& V, 
        optix::Onb onb, 
        float3& attenuation, float3& direction, float& pdfSolid)
{
    const float z1 = rnd( seed );
    const float z2 = rnd( seed );
    const float z = rnd( seed );

    float alpha = rough * rough;
    float k = (alpha + 2 * rough + 1) / 8.0;
    float alpha2 = alpha * alpha;
    
    float3 L;
    if(z < 0.5){
        cosine_sample_hemisphere(z1, z2, L);
        onb.inverse_transform(L);
        direction = L;
        attenuation =  2 * attenuation * albedoValue;
    }
    else{
        // Compute the half angle 
        float phi = 2 * M_PIf * z1;
        float cosTheta = sqrt( (1 - z2) / (1 + (alpha2 - 1) * z2) );
        float sinTheta = sqrt( 1 - cosTheta * cosTheta);

        float3 H = make_float3(
                sinTheta * cos(phi),
                sinTheta * sin(phi),
                cosTheta);
        onb.inverse_transform(H);
        L = 2 * dot(V, H) * H - V;
        direction = L;

        float NoV = fmaxf(dot(N, V), 0.0000001f);
        float NoL = dot(N, L);
        float NoH = fmaxf(dot(N, H), 0.00000001f);
        float VoH = fmaxf(dot(V, H), 0.00000001f);

        if( NoL >= 0){
            float G1 = NoV / (NoV * (1-k) + k);
            float G2 = NoL / (NoL * (1-k) + k);
            float FMi = (-5.55473 * VoH - 6.98316) * VoH;
            float3 F = fresnel + (1 - fresnel) * pow(2.0f, FMi);
            float3 reflec = F * G1 * G2 * VoH / NoH / NoV;

            attenuation = 2 * attenuation * reflec;
        }
        else{
            attenuation = make_float3(0.0f);
        }
    }
    pdfSolid = pdf(L, V, N, rough);
}


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
 
    float3 albedoValue;
    if(isAlbedoTexture == 0){
        albedoValue = albedo;
    }
    else{
        albedoValue = make_float3(tex2D(albedoMap, texcoord.x, texcoord.y) );
        albedoValue.x = pow(albedoValue.x, 2.2);
        albedoValue.y = pow(albedoValue.y, 2.2);
        albedoValue.z = pow(albedoValue.z, 2.2);
    }

    float roughValue = (isRoughTexture == 0) ? rough :
        tex2D(roughMap, texcoord.x, texcoord.y).x;
    
    float metallicValue = (isMetallicTexture == 0) ? metallic :
        tex2D(metallicMap, texcoord.x, texcoord.y).x;

    float3 fresnel = F0 * (1 - metallicValue) + metallicValue * albedoValue;
    albedoValue = (1 - metallicValue) * albedoValue;
    
    float3 V = normalize(-ray.direction );    
    
    float3 N;
    if( isNormalTexture == 0){
        N = ffnormal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x, texcoord.y) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * ffnormal;
    }
    N = normalize(N );
    if(dot(N, V) < 0)
        N = -N;
    optix::Onb onb(N );
 
    float3 hitPoint = ray.origin + t_hit * ray.direction;
    prd_radiance.origin = hitPoint;

    // Connect to the area Light
    {
        if(isAreaLight == 1){
            float3 position, radiance, normal;
            float pdfAreaLight;
            sampleAreaLight(prd_radiance.seed, radiance, position, normal, pdfAreaLight);
   
            float Dist = length(position - hitPoint);
            float3 L = normalize(position - hitPoint);

            if(dot(N, L) * dot(N, V) > 0 ){
                float cosPhi = dot(L, normal);
                cosPhi = (cosPhi < 0) ? -cosPhi : cosPhi;

                Ray shadowRay = make_Ray(hitPoint, L, 1, scene_epsilon, Dist - scene_epsilon);
                PerRayData_shadow prd_shadow; 
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, N, roughValue, fresnel, V, L, radiance) * cosPhi / Dist / Dist;
                    float pdfSolidBRDF = pdf(L, V, N, roughValue);
                    float pdfAreaBRDF = pdfSolidBRDF * cosPhi / Dist / Dist;

                    float pdfAreaLight2 = pdfAreaLight * pdfAreaLight;
                    float pdfAreaBRDF2 = pdfAreaBRDF * pdfAreaBRDF;

                    prd_radiance.radiance += intensity * pdfAreaLight / (pdfAreaBRDF2 + pdfAreaLight2) * prd_radiance.attenuation;
                }
            }
        }
    }
    
    // Connect to point light 
    {
        if(isPointLight == 1){
            // Connect to every point light 
            for(int i = 0; i < pointLightNum; i++){
                float3 position = pointLights[i].position;
                float3 radiance = pointLights[i].intensity;
                float3 L = normalize(position - hitPoint);
                float Dist = length(position - hitPoint);

                if(dot(N, L) * dot(N, V) > 0 ){
                    Ray shadowRay = make_Ray(hitPoint + 0.1 * L * scene_epsilon, L, 1, scene_epsilon, Dist - scene_epsilon);
                    PerRayData_shadow prd_shadow; 
                    prd_shadow.inShadow = false;
                    rtTrace(top_object, shadowRay, prd_shadow);
                    if(prd_shadow.inShadow == false){
                        float3 intensity = evaluate(albedoValue, N, roughValue, fresnel, V, L, radiance) / Dist/ Dist;
                        prd_radiance.radiance += intensity * prd_radiance.attenuation;
                    }
                }
            }
        }
    }

    // Connect to the environmental map 
    { 
        if(isEnvmap == 1){
            float3 L, radiance;
            float pdfSolidEnv;
            sampleEnvironmapLight(prd_radiance.seed, radiance, L, pdfSolidEnv);

            if( dot(L, N) * dot(V, N) > 0){
                Ray shadowRay = make_Ray(hitPoint + 0.1 * scene_epsilon, L, 1, scene_epsilon, infiniteFar);
                PerRayData_shadow prd_shadow;
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, N, roughValue, fresnel, V, L, radiance);
                    float pdfSolidBRDF = pdf(L, V, N, roughValue);
                    float pdfSolidBRDF2 = pdfSolidBRDF * pdfSolidBRDF;
                    float pdfSolidEnv2 = pdfSolidEnv * pdfSolidEnv;
                    prd_radiance.radiance += intensity * pdfSolidEnv / (pdfSolidEnv2 + pdfSolidBRDF2) * prd_radiance.attenuation; 
                }
            }
        }
    }

    // Sammple the new ray 
    sample(prd_radiance.seed, 
        albedoValue, N, roughValue, fresnel, V, 
        onb, 
        prd_radiance.attenuation, prd_radiance.direction, prd_radiance.pdf);
}

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

